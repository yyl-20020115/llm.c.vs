/*
Kernels for layernorm forward pass.

Compile example:
nvcc -O3 --use_fast_math layernorm_forward.cu -o layernorm_forward

version 1 is naive port from CPU code to kernel: parallelizes over B,T, loops over C
./layernorm_forward 1

version 2 parallelizes over all of B,T,C
./layernorm_forward 2
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// ----------------------------------------------------------------------------
// CUDA utils

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

// error checking
void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};
#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

// ----------------------------------------------------------------------------
// CPU code reference

// GPT-2 layernorm forward pass
void layernorm_forward_cpu(float* out, float* mean, float* rstd,
                       float* inp, float* weight, float* bias,
                       int B, int T, int C) {
    float eps = 1e-5f;
    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            // seek to the input position inp[b,t,:]
            float* x = inp + b * T * C + t * C;
            // calculate the mean
            float m = 0.0f;
            for (int i = 0; i < C; i++) {
                m += x[i];
            }
            m = m/C;
            // calculate the variance (without any bias correction)
            float v = 0.0f;
            for (int i = 0; i < C; i++) {
                float xshift = x[i] - m;
                v += xshift * xshift;
            }
            v = v/C;
            // calculate the rstd
            float s = 1.0f / sqrtf(v + eps);
            // seek to the output position in out[b,t,:]
            float* out_bt = out + b * T * C + t * C;
            for (int i = 0; i < C; i++) {
                float n = (s * (x[i] - m)); // normalized output
                float o = n * weight[i] + bias[i]; // scale and shift it
                out_bt[i] = o; // write
            }
            // cache the mean and rstd for the backward pass later
            mean[b * T + t] = m;
            rstd[b * T + t] = s;
        }
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

// naive drag and drop implementation into kernel, parallelize over B,T, loop over C
__global__ void layernorm_forward_kernel1(float* out, float* mean, float* rstd,
                                 float* inp, float* weight, float* bias,
                                 int N, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float eps = 1e-5f;

    if (idx < N) {
        // seek to the input position inp[idx,:]
        float* x = inp + idx * C;
        // calculate the mean
        float m = 0.0f;
        for (int i = 0; i < C; i++) {
            m += x[i];
        }
        m = m / C;
        // calculate the variance (without any bias correction)
        float v = 0.0f;
        for (int i = 0; i < C; i++) {
            float xshift = x[i] - m;
            v += xshift * xshift;
        }
        v = v / C;
        // calculate the rstd
        float s = 1.0f / sqrtf(v + eps);
        // seek to the output position in out[idx,:]
        float* out_idx = out + idx * C;
        for (int i = 0; i < C; i++) {
            float n = (s * (x[i] - m)); // normalized output
            float o = n * weight[i] + bias[i]; // scale and shift it
            out_idx[i] = o; // write
        }
        // cache the mean and rstd for the backward pass later
        mean[idx] = m;
        rstd[idx] = s;
    }
}

__global__ void mean_kernel(float* mean, float* inp, int N, int C, int block_size) {
    extern __shared__ float shared[];
    int idx = blockIdx.x; // range [0, B*T)
    int tid = threadIdx.x; // range [0, block_size)
    float* x = inp + idx * C;
    // thread coarsening
    float sum = 0.0f;
    for (int i = tid; i < C; i += block_size) {
        sum += x[i];
    }
    shared[tid] = sum;
    __syncthreads();
    // reductions
    for (int stride = block_size / 2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (tid < stride) {
            shared[tid] += shared[tid + stride];
        }
    }
    // write the final result (at thread 0) to global memory
    if (tid == 0) {
        mean[idx] = shared[0] / C;
    }
}

__global__ void rstd_kernel(float* rstd, float* inp, float* mean, int N, int C, int block_size) {
    extern __shared__ float shared[];
    int idx = blockIdx.x; // range [0, B*T)
    int tid = threadIdx.x; // range [0, block_size)
    float* x = inp + idx * C;
    float m = mean[idx];
    // thread coarsening
    float sum = 0.0f;
    for (int i = tid; i < C; i += block_size) {
        float diff = x[i] - m;
        sum += diff * diff;
    }
    shared[tid] = sum;
    __syncthreads();
    // reductions
    for (int stride = block_size / 2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (tid < stride) {
            shared[tid] += shared[tid + stride];
        }
    }
    // write the final result (at thread 0) to global memory
    if (tid == 0) {
        rstd[idx] = 1.0f / sqrtf(shared[0] / C + 1e-5f);
    }
}

__global__ void normalization_kernel(float* out, float* inp, float* mean, float* rstd,
                                     float* weight, float* bias, int B, int T, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int bt = idx / C;
    int c = idx % C;

    float m = mean[bt];
    float s = rstd[bt];
    float xi = inp[idx];
    float n = s * (xi - m);
    float o = n * weight[c] + bias[c];

    out[idx] = o;
}

// ----------------------------------------------------------------------------
// kernel launcher

void layernorm_forward1(float* out, float* mean, float* rstd,
                           float* inp, float* weight, float* bias,
                           int B, int T, int C,
                           const int block_size) {
    const int N = B * T;
    const int grid_size = CEIL_DIV(N, block_size);
    layernorm_forward_kernel1<<<grid_size, block_size>>>(out, mean, rstd, inp, weight, bias, N, C);
    cudaCheck(hipGetLastError());
}

void layernorm_forward2(float* out, float* mean, float* rstd,
                       float* inp, float* weight, float* bias,
                       int B, int T, int C,
                       const int block_size) {
    int N = B * T;
    // in mean and rstd, threads cooperate within blocks via reductions
    mean_kernel<<<B * T, block_size, block_size * sizeof(float)>>>(mean, inp, N, C, block_size);
    cudaCheck(hipGetLastError());
    rstd_kernel<<<B * T, block_size, block_size * sizeof(float)>>>(rstd, inp, mean, N, C, block_size);
    cudaCheck(hipGetLastError());
    // in the normalization, everything just gets flattened out
    const int block_size2 = 256;
    const int grid_size = CEIL_DIV(B * T * C, block_size2);
    normalization_kernel<<<grid_size, block_size2>>>(out, inp, mean, rstd, weight, bias, B, T, C);
    cudaCheck(hipGetLastError());
}

// kernel version dispatch
void layernorm_forward(int kernel_num,
                    float* out, float* mean, float* rstd,
                    float* inp, float* weight, float* bias,
                    int B, int T, int C,
                    const int block_size) {
    switch (kernel_num) {
        case 1:
            layernorm_forward1(out, mean, rstd, inp, weight, bias, B, T, C, block_size);
            break;
        case 2:
            layernorm_forward2(out, mean, rstd, inp, weight, bias, B, T, C, block_size);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}

// ----------------------------------------------------------------------------
// random utils

float* make_random_float(int N) {
    float* arr = (float*)malloc(N * sizeof(float));
    for (int i = 0; i < N; i++) {
        arr[i] = ((float)rand() / RAND_MAX) * 2.0 - 1.0;
    }
    return arr;
}

// ----------------------------------------------------------------------------

int main(int argc, char **argv) {
    srand(0);

    int B = 8;
    int T = 1024;
    int C = 768;

    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));

    // create host memory of random numbers
    float* out = (float*)malloc(B * T * C * sizeof(float));
    float* mean = (float*)malloc(B * T * sizeof(float));
    float* rstd = (float*)malloc(B * T * sizeof(float));
    float* inp = make_random_float(B * T * C);
    float* weight = make_random_float(C);
    float* bias = make_random_float(C);

    // move to GPU
    float* d_out;
    float* d_mean;
    float* d_rstd;
    float* d_inp;
    float* d_weight;
    float* d_bias;
    cudaCheck(hipMalloc(&d_out, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_mean, B * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_rstd, B * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_weight, C * sizeof(float)));
    cudaCheck(hipMalloc(&d_bias, C * sizeof(float)));
    cudaCheck(hipMemcpy(d_inp, inp, B * T * C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_weight, weight, C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_bias, bias, C * sizeof(float), hipMemcpyHostToDevice));

    // read kernel_num from command line
    int kernel_num = 2;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    // first check the correctness of the kernel
    layernorm_forward_cpu(out, mean, rstd, inp, weight, bias, B, T, C);
    layernorm_forward(kernel_num, d_out, d_mean, d_rstd, d_inp, d_weight, d_bias, B, T, C, 256);
    float* out_gpu = (float*)malloc(B * T * C * sizeof(float));
    cudaCheck(hipMemcpy(out_gpu, d_out, B * T * C * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < B * T * C; i++) {
        // print the first few comparisons
        if (i < 5) {
            printf("%f %f\n", out[i], out_gpu[i]);
        }
        // ensure correctness for all elements
        if (fabs(out[i] - out_gpu[i]) > 1e-5) {
            printf("Mismatch at %d: %f vs %f\n", i, out[i], out_gpu[i]);
            exit(1);
        }
    }
    printf("Results match at block_size=256!\n");

    // time the kernel at different block sizes
    int block_sizes[] = {32, 64, 128, 256, 512, 1024};

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];

        int repeat_times = 1000;
        hipEvent_t start, stop;
        cudaCheck(hipEventCreate(&start));
        cudaCheck(hipEventCreate(&stop));
        cudaCheck(hipEventRecord(start, 0));
        for (int i = 0; i < repeat_times; i++) {
            layernorm_forward(kernel_num, d_out, d_mean, d_rstd, d_inp, d_weight, d_bias, B, T, C, block_size);
        }
        cudaCheck(hipEventRecord(stop, 0));
        cudaCheck(hipEventSynchronize(start));
        cudaCheck(hipEventSynchronize(stop));
        float elapsed_time;
        cudaCheck(hipEventElapsedTime(&elapsed_time, start, stop));

        // napkin math: estimate the memory bandwidth achieved
        // e.g. A100 40GB PCIe is advertised at 1,555GB/s
        long memory_ops = (2 * B * T * C) * 4; // *4 for float
        float memory_bandwidth = memory_ops / (elapsed_time / repeat_times) / 1e6;

        printf("block_size %4d | time %f ms | bandwidth %f GB/s\n", block_size, elapsed_time / repeat_times, memory_bandwidth);
    }

    // free memory
    free(out);
    free(mean);
    free(rstd);
    free(inp);
    free(weight);
    free(bias);
    free(out_gpu);
    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_mean));
    cudaCheck(hipFree(d_rstd));
    cudaCheck(hipFree(d_inp));
    cudaCheck(hipFree(d_weight));
    cudaCheck(hipFree(d_bias));

    return 0;
}