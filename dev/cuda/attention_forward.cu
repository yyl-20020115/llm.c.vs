/*
Kernels for attention forward pass.

Compile example:
nvcc -O3 --use_fast_math attention_forward.cu -o attention_forward -lcublas

version 1 is naive port from CPU code to kernel, parallelize over batch, time, heads only
./attention_forward 1

version 2 is a naive implementation of flash attention, taken, adapted from
https://github.com/tspeterkim/flash-attention-minimal
and with help from
https://github.com/leloykun/flash-hyperbolic-attention-minimal
sadly, this flash attention version seems about 3X slower than the naive version
./attention_forward 2

version 3 is a cuBLAS + softmax version, similar to the PyTorch implementation
cuBLAS is used both to calculate the QK^T and the final weighted sum
the softmax is calculated using a custom, efficient kernel as well
this turns out to be ~20X faster than (1) nice
./attention_forward 3
*/

#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

// ----------------------------------------------------------------------------
// CUDA utils

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

// error checking
void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};
#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

// ----------------------------------------------------------------------------
// CPU code reference

void attention_forward_cpu(float* out, float* preatt, float* att,
                       float* inp,
                       int B, int T, int C, int NH) {
    // input is (B, T, 3C) Q,K,V
    // preatt, att are (B, NH, T, T)
    // output is (B, T, C)
    int C3 = C*3;
    int hs = C / NH; // head size
    float scale = 1.0 / sqrtf(hs);

    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            for (int h = 0; h < NH; h++) {
                float* query_t = inp + b * T * C3 + t * C3 + h * hs;
                float* preatt_bth = preatt + b*NH*T*T + h*T*T + t*T;
                float* att_bth = att + b*NH*T*T + h*T*T + t*T;

                // pass 1: calculate query dot key and maxval
                float maxval = -10000.0f; // TODO something better
                for (int t2 = 0; t2 <= t; t2++) {
                    float* key_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C; // +C because it's key

                    // (query_t) dot (key_t2)
                    float val = 0.0f;
                    for (int i = 0; i < hs; i++) {
                        val += query_t[i] * key_t2[i];
                    }
                    val *= scale;
                    if (val > maxval) {
                        maxval = val;
                    }

                    preatt_bth[t2] = val;
                }
                // pad with -INFINITY outside of autoregressive region for debugging comparisons
                for (int t2 = t+1; t2 < T; t2++) {
                    preatt_bth[t2] = -INFINITY;
                }

                // pass 2: calculate the exp and keep track of sum
                float expsum = 0.0f;
                for (int t2 = 0; t2 <= t; t2++) {
                    float expv = expf(preatt_bth[t2] - maxval);
                    expsum += expv;
                    att_bth[t2] = expv;
                }
                float expsum_inv = expsum == 0.0f ? 0.0f : 1.0f / expsum;

                // pass 3: normalize to get the softmax
                for (int t2 = 0; t2 < T; t2++) {
                    if (t2 <= t) {
                        att_bth[t2] *= expsum_inv;
                    } else {
                        // causal attention mask. not strictly necessary to set to zero here
                        // only doing this explicitly for debugging and checking to PyTorch
                        att_bth[t2] = 0.0f;
                    }
                }

                // pass 4: accumulate weighted values into the output of attention
                float* out_bth = out + b * T * C + t * C + h * hs;
                for (int i = 0; i < hs; i++) { out_bth[i] = 0.0f; }
                for (int t2 = 0; t2 <= t; t2++) {
                    float* value_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C*2; // +C*2 because it's value
                    float att_btht2 = att_bth[t2];
                    for (int i = 0; i < hs; i++) {
                        out_bth[i] += att_btht2 * value_t2[i];
                    }
                }
            }
        }
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

__global__ void attention_query_key_kernel1(float* preatt, float* inp,
                                           int B, int T, int C, int NH) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = B * NH * T * T;

    if (idx < total_threads) {
        int t2 = idx % T;
        int t = (idx / T) % T;
        if (t2 > t) {
            // autoregressive mask
            preatt[idx] = -INFINITY;
            return;
        }
        int h = (idx / (T * T)) % NH;
        int b = idx / (NH * T * T);

        int C3 = C*3;
        int hs = C / NH; // head size
        float* query_t = inp + b * T * C3 + t * C3 + h * hs;
        float* key_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C; // +C because it's key

        // (query_t) dot (key_t2)
        float val = 0.0f;
        for (int i = 0; i < hs; i++) {
            val += query_t[i] * key_t2[i];
        }
        val *= 1.0 / sqrtf(hs);

        preatt[idx] = val;
    }
}

__global__ void attention_softmax_kernel1(float* att, float* preatt,
                                         int B, int T, int NH) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = B * T * NH;

    if (idx < total_threads) {
        int h = idx % NH;
        int t = (idx / NH) % T;
        int b = idx / (NH * T);

        float* preatt_bth = preatt + b*NH*T*T + h*T*T + t*T;
        float* att_bth = att + b*NH*T*T + h*T*T + t*T;

        // find maxval
        float maxval = -10000.0f; // TODO something better
        for (int t2 = 0; t2 <= t; t2++) {
            if (preatt_bth[t2] > maxval) {
                maxval = preatt_bth[t2];
            }
        }

        // calculate the exp and keep track of sum
        float expsum = 0.0f;
        for (int t2 = 0; t2 <= t; t2++) {
            float expv = expf(preatt_bth[t2] - maxval);
            expsum += expv;
            att_bth[t2] = expv;
        }
        float expsum_inv = expsum == 0.0f ? 0.0f : 1.0f / expsum;

        // normalize to get the softmax
        for (int t2 = 0; t2 < T; t2++) {
            if (t2 <= t) {
                att_bth[t2] *= expsum_inv;
            } else {
                // causal attention mask. not strictly necessary to set to zero here
                // only doing this explicitly for debugging and checking to PyTorch
                att_bth[t2] = 0.0f;
            }
        }
    }
}

// warp-level reduction for finding the maximum value
__device__ float warpReduceMax(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val = fmaxf(val, __shfl_down_sync(0xFFFFFFFF, val, offset));
    }
    return val;
}

// warp-level reduction for summing values
__device__ float warpReduceSum(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    }
    return val;
}

__global__ void softmax_forward_kernel4(float* out, float* inp, int N, int C) {
    // out is (N, C) just like inp. Each row of inp will get softmaxed.
    // same as kernel3, but can handle any block size (multiple of 32)
    // each row of C elements is handled by block_size threads
    // furthermore, each block_size threads get executed in warps of 32 threads

    // special reduction operations warpReduceMax/warpReduceSum are used for intra-warp reductions
    // shared memory is used for inter-warp reduction
    extern __shared__ float shared[];
    int idx = blockIdx.x;
    int tid = threadIdx.x;
    int warpId = threadIdx.x / 32; // warp index within a block
    int laneId = threadIdx.x % 32; // thread index within a warp

    // the number of warps per block. recall that blockDim.x is block_size
    int warpsPerBlock = blockDim.x / 32;

    // shared[] must be allocated to have 2 * warpsPerBlock elements
    // first half for max values, the second half for sum values
    float* maxvals = shared;
    float* sumvals = &shared[warpsPerBlock];

    // one row of inp, i.e. inp[idx, :] of shape (C,)
    float* x = inp + idx * C;

    // first, thread coarsening by directly accessing global memory in series
    float maxval = -INFINITY;
    for (int i = tid; i < C; i += blockDim.x) {
        maxval = fmaxf(maxval, x[i]);
    }
    // now within-warp reductions for maxval
    maxval = warpReduceMax(maxval);

    // the 0th thread of each warp writes the maxval of that warp to shared memory
    if (laneId == 0) maxvals[warpId] = maxval;
    __syncthreads();

    // now the 0th thread reduces the maxvals in shared memory, i.e. across warps
    if (tid == 0) {
        float val = maxvals[tid];
        for (int i = 1; i < warpsPerBlock; i++) {
            val = fmaxf(val, maxvals[i]);
        }
        // store the final max in the first position
        maxvals[0] = val;
    }
    __syncthreads();
    // broadcast the max to all threads
    float offset = maxvals[0];

    // compute expf and write the result to global memory
    for (int i = tid; i < C; i += blockDim.x) {
        // subtract max for numerical stability
        out[idx * C + i] = expf(x[i] - offset);
    }

    // okay now we calculated exp(x - max(x))
    // step 2: sum all the values and divide by the sum

    // thread coarsening for sum
    x = out + idx * C;
    float sumval = 0.0f;
    for (int i = tid; i < C; i += blockDim.x) {
        sumval += x[i];
    }
    // within-warp reduction for sumval
    sumval = warpReduceSum(sumval);

    // write sumval to shared memory
    if (laneId == 0) sumvals[warpId] = sumval;
    __syncthreads();

    // inter-thread reduction of sum
    if (tid == 0) {
        float val = sumvals[tid];
        for (int i = 1; i < warpsPerBlock; ++i) {
            val += sumvals[i];
        }
        sumvals[0] = val;
    }
    __syncthreads();
    // broadcast the sum to all threads
    float sum = sumvals[0];

    // divide the whole row by the sum
    for (int i = tid; i < C; i += blockDim.x) {
        out[idx * C + i] = x[i] / sum;
    }
}

__global__ void attention_value_kernel1(float* out, float* att, float* inp,
                                       int B, int T, int C, int NH) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = B * T * NH;

    if (idx < total_threads) {
        int h = idx % NH;
        int t = (idx / NH) % T;
        int b = idx / (NH * T);

        int C3 = C*3;
        int hs = C / NH; // head size

        float* out_bth = out + b * T * C + t * C + h * hs;
        float* att_bth = att + b*NH*T*T + h*T*T + t*T;

        for (int i = 0; i < hs; i++) { out_bth[i] = 0.0f; }
        for (int t2 = 0; t2 <= t; t2++) {
            float* value_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C*2; // +C*2 because it's value
            float att_btht2 = att_bth[t2];
            for (int i = 0; i < hs; i++) {
                out_bth[i] += att_btht2 * value_t2[i];
            }
        }
    }
}

__global__
void attention_forward_kernel2(
    const float* Q,
    const float* K,
    const float* V,
    const int N,
    const int d,
    const int Tc,
    const int Tr,
    const int Bc,
    const int Br,
    const float softmax_scale,
    float* l,
    float* m,
    float* O
) {
    int tx = threadIdx.x;
    int bx = blockIdx.x; int by = blockIdx.y;  // batch and head index

    // Offset into Q,K,V,O,l,m - different for each batch and head
    int qkv_offset = (bx * gridDim.y * N * d) + (by * N * d);  // gridDim.y = nh
    int lm_offset = (bx * gridDim.y * N) + (by * N);  // offset for l and m

    // Define SRAM for Q,K,V,S
    extern __shared__ float sram[];
    int tile_size = Bc * d;  // size of Qi, Kj, Vj
    float* Qi = sram;
    float* Kj = &sram[tile_size];
    float* Vj = &sram[tile_size * 2];
    float* S = &sram[tile_size * 3];

    for (int j = 0; j < Tc; j++) {

        // Load Kj, Vj to SRAM
        for (int x = 0; x < d; x++) {
            Kj[(tx * d) + x] = K[qkv_offset + (tile_size * j) + (tx * d) + x];
            Vj[(tx * d) + x] = V[qkv_offset + (tile_size * j) + (tx * d) + x];
        }
        __syncthreads();  // such that the inner loop can use the correct Kj, Vj

        for (int i = 0; i < Tr; i++)  {
            // if past the end of the sequence, break
            if (i * Br + tx >= N) {
                break;
            }

            // Load Qi to SRAM, l and m to registers
            for (int x = 0; x < d; x++) {
                Qi[(tx * d) + x] = Q[qkv_offset + (tile_size * i) + (tx * d) + x];
            }
            float row_m_prev = m[lm_offset + (Br * i) + tx];
            float row_l_prev = l[lm_offset + (Br * i) + tx];

            // S = QK^T, row_m = rowmax(S)
            // S[tx][y] = Sum_{x = 0}^{d-1} {Qi[tx][x] * Kj[y][x]}
            // row_m = Max_{y = 0}^{Bc-1} S[tx][y]
            // with causal masking
            float row_m = -INFINITY;
            for (int y = 0; y < Bc; y++) {
                if (j * Bc + y >= N) {
                    break;
                }
                float sum = 0;
                for (int x = 0; x < d; x++) {
                    sum += Qi[(tx * d) + x] * Kj[(y * d) + x];
                }
                sum *= softmax_scale;
                if (i * Br + tx < j * Bc + y)
                    sum = -INFINITY;
                S[(Bc * tx) + y] = sum;

                if (sum > row_m)
                    row_m = sum;
            }

            // implement softmax with causal masking
            // P = exp(S - row_m), row_l = rowsum(P)
            // P[tx][y] = exp(S[tx][y] - row_m)
            float row_l = 0;
            for (int y = 0; y < Bc; y++) {
                if (j * Bc + y >= N) {
                    break;
                }
                if (i * Br + tx < j * Bc + y)
                    S[(Bc * tx) + y] = 0;
                else
                    S[(Bc * tx) + y] = __expf(S[(Bc * tx) + y] - row_m);
                row_l += S[(Bc * tx) + y];
            }

            // Compute new m and l
            float row_m_new = max(row_m_prev, row_m);
            float row_l_new = (__expf(row_m_prev - row_m_new) * row_l_prev) + (__expf(row_m - row_m_new) * row_l);

            // Write O, l, m to HBM
            for (int x = 0; x < d; x++) {
                float pv = 0;  // Pij * Vj
                for (int y = 0; y < Bc; y++) {
                    if (j * Bc + y >= N) {
                        break;
                    }
                    pv += S[(Bc * tx) + y] * Vj[(y * d) + x];
                }
                O[qkv_offset + (tile_size * i) + (tx * d) + x] = (1 / row_l_new) \
                    * ((row_l_prev * __expf(row_m_prev - row_m_new) * O[qkv_offset + (tile_size * i) + (tx * d) + x]) \
                    + (__expf(row_m - row_m_new) * pv));
            }
            m[lm_offset + (Br * i) + tx] = row_m_new;
            l[lm_offset + (Br * i) + tx] = row_l_new;
        }
        __syncthreads();  // otherwise, thread can use the wrong Kj, Vj in inner loop
    }
}

__global__ void permute_kernel(float* q, float* k, float* v,
                               const float* inp,
                               int B, int N, int NH, int d) {
    // okay so now, this kernel wants Q,K,V to all be of shape (B, NH, N, d)
    // but instead, we have a single tensor QKV (inp) of shape (B, N, 3, NH, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Q[b][nh_][n][d_] = inp[b][n][0][nh_][d_]

    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;

        int inp_idx = \
            (b * N * 3 * NH * d)
            +   (n * 3 * NH * d)
            +       (0 * NH * d)
            +          (nh_ * d)
            +                d_;

        q[idx] = inp[inp_idx];
        k[idx] = inp[inp_idx + NH * d];
        v[idx] = inp[inp_idx + 2 * (NH * d)];
    }
}

__global__ void unpermute_kernel(float* inp, float *out, int B, int N, int NH, int d) {
   // out has shape (B, nh, N, d) but we need to unpermute it to (B, N, nh, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // out[b][n][nh_][d_] <- inp[b][nh_][n][d_]
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;

        int other_idx = (b * NH * N * d) + (n * NH * d) + (nh_ * d) + d_;
        out[other_idx] = inp[idx];
    }
}

__global__ void scale_kernel(float* inp, float scale, int B, int NH, int T) {
    // scales the pre-softmax attention scores by scale
    // and sets the autoregressive locations to -INFINITY
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < B * NH * T * T) {
        int rest = idx % (NH * T * T);
        rest = rest % (T * T);
        int t2 = rest / T;
        int t = rest % T;
        if (t > t2) {
            inp[idx] = -INFINITY;
        } else {
            inp[idx] *= scale;
        }
    }
}

// ----------------------------------------------------------------------------
// kernel launcher

void attention_forward1(float* out, float* preatt, float* att,
                       float* inp,
                       int B, int T, int C, int NH,
                       const int block_size) {
    // attention calculation
    int total_threads = B * NH * T * T;
    int num_blocks = CEIL_DIV(total_threads, block_size);
    attention_query_key_kernel1<<<num_blocks, block_size>>>(preatt, inp, B, T, C, NH);
    // softmax and value accumulation
    total_threads = B * T * NH;
    num_blocks = CEIL_DIV(total_threads, block_size);
    attention_softmax_kernel1<<<num_blocks, block_size>>>(att, preatt, B, T, NH);
    attention_value_kernel1<<<num_blocks, block_size>>>(out, att, inp, B, T, C, NH);
}


void attention_forward2(float* out,
                       float* inp,
                       int B, int T, int C, int NH,
                       const int block_size) {
    // TODO there should be no mallocs inside any of these functions!
    // not fixing this because we don't intend to use attention_forward2,
    // it seems to be way too slow as is

    // these are hardcoded to 32 for now
    const int Bc = 32;
    const int Br = 32;
    // renaming these to be consistent with the kernel
    // const int B = B;
    const int nh = NH;
    const int N = T;
    const int d = C / NH;
    // more
    const int Tc = ceil((float) N / Bc);
    const int Tr = ceil((float) N / Br);
    const float softmax_scale = 1.0 / sqrt(d);
    // create some temporary memory
    float* l;
    float* m;
    cudaCheck(hipMalloc(&l, B * nh * N * sizeof(float)));
    cudaCheck(hipMalloc(&m, B * nh * N * sizeof(float)));
    cudaCheck(hipMemset(l, 0, B * nh * N * sizeof(float)));
    cudaCheck(hipMemset(m, -10000.0f, B * nh * N * sizeof(float)));

    // calculate SRAM size needed per block, ensure we have enough shared memory
    int col_tile_size = Bc * d;  // size of Kj, Vj
    int row_tile_size = Br * d;  // size of Qi
    const int sram_size =
        (2 * col_tile_size * sizeof(float))  // SRAM size for Kj, Vj
        + (row_tile_size * sizeof(float))  // SRAM size for Qi
        + (Bc * Br * sizeof(float));  // SRAM size for S
    int max_sram_size;
    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    if (sram_size > max_sram_size) {
        printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, sram_size);
        printf("SRAM size exceeds maximum shared memory per block\n");
        printf("Try decreasing col_tile_size or row_tile_size further\n");
        exit(1);
    }

    // grid and block dims
    dim3 grid_dim(B, nh);  // batch_size x num_heads
    dim3 block_dim(Br);  // Br threads per block

    // okay so now, this kernel wants Q,K,V to all be of shape (B, nh, N, d)
    // but instead, we have a single tensor QKV (inp) of shape (B, N, 3, nh, d)
    // so we have to permute the tensor using a kernel with block_size
    float *q, *k, *v;
    cudaCheck(hipMalloc(&q, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&k, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&v, B * T * C * sizeof(float)));
    int total_threads = B * N * nh * d;
    int num_blocks = CEIL_DIV(total_threads, block_size);
    permute_kernel<<<num_blocks, block_size>>>(q, k, v, inp, B, N, nh, d);

    // now actually call the flash attention kernel
    attention_forward_kernel2<<<grid_dim, block_dim, sram_size>>>(
        q, k, v,
        N, d, Tc, Tr, Bc, Br, softmax_scale,
        l, m, out
    );

    // out has shape (B, nh, N, d) but we need to unpermute it to (B, N, nh, d)
    unpermute_kernel<<<num_blocks, block_size>>>(out, q, B, N, nh, d);
    cudaCheck(hipMemcpy(out, q, B * T * C * sizeof(float), hipMemcpyDeviceToDevice));

    // free memory
    cudaCheck(hipFree(l));
    cudaCheck(hipFree(m));
    cudaCheck(hipFree(q));
    cudaCheck(hipFree(k));
    cudaCheck(hipFree(v));
}

void attention_forward3(float* out, float* vaccum, float* qkvr, float* preatt, float* att,
                       float* inp,
                       int B, int T, int C, int NH,
                       const int block_size) {
    // inp is (B, T, 3C) QKV
    // preatt, att are (B, NH, T, T)
    // output is (B, T, C)
    int HS = C / NH; // head size

    // permute and separate inp from (B, T, 3, NH, HS) to 3X (B, NH, T, HS)
    float *q, *k, *v;
    q = qkvr + 0 * B * T * C;
    k = qkvr + 1 * B * T * C;
    v = qkvr + 2 * B * T * C;
    int total_threads = B * NH * T * HS;
    int num_blocks = CEIL_DIV(total_threads, block_size);
    permute_kernel<<<num_blocks, block_size>>>(q, k, v, inp, B, T, NH, HS);

    // batched matrix multiply with cuBLAS
    hipblasHandle_t handle;
    hipblasStatus_t stat = hipblasCreate(&handle);
    const float alpha = 1.0f;
    const float beta = 0.0f;
    stat = hipblasSgemmStridedBatched(handle,
                            HIPBLAS_OP_T, HIPBLAS_OP_N,
                            T, T, HS,
                            &alpha,
                            k, HS, T * HS,
                            q, HS, T * HS,
                            &beta,
                            preatt, T, T * T,
                            B * NH);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("hipblasSgemm failed\n");
        exit(1);
    }

    // multiply all elements of preatt elementwise by scale
    float scale = 1.0 / sqrtf(HS);
    total_threads = B * NH * T * T;
    num_blocks = CEIL_DIV(total_threads, block_size);
    scale_kernel<<<num_blocks, block_size>>>(preatt, scale, B, NH, T);

    // softmax. preatt is (B, NH, T, T) but we view it as (B * NH * T, T) and use the softmax kernel
    int softmax_block_size = 256;
    int grid_size = B * NH * T;
    size_t shared_mem_size = 2 * softmax_block_size / 32 * sizeof(float);
    softmax_forward_kernel4<<<grid_size, softmax_block_size, shared_mem_size>>>(att, preatt, B * NH * T, T);

    // new approach: first cuBLAS another batched matmul
    // y = att @ v # (B, nh, T, T) @ (B, nh, T, hs) -> (B, nh, T, hs)
    stat = hipblasSgemmStridedBatched(handle,
                            HIPBLAS_OP_N, HIPBLAS_OP_N,
                            HS, T, T,
                            &alpha,
                            v, HS, T * HS,
                            att, T, T * T,
                            &beta,
                            vaccum, HS, T * HS,
                            B * NH);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("hipblasSgemm failed\n");
        exit(1);
    }

    // now unpermute
    // y = y.transpose(1, 2).contiguous().view(B, T, C) # re-assemble all head outputs side by side
    num_blocks = CEIL_DIV(B * T * C, block_size);
    unpermute_kernel<<<num_blocks, block_size>>>(vaccum, out, B, T, NH, HS);

    // cleanups
    hipblasDestroy(handle);
}

// kernel version dispatch
void attention_forward(int kernel_num,
                       float* out, float* vaccum, float* qkvr, float* preatt, float* att,
                       float* inp,
                       int B, int T, int C, int NH,
                       const int block_size) {
    switch (kernel_num) {
        case 1:
            attention_forward1(out, preatt, att, inp, B, T, C, NH, block_size);
            break;
        case 2:
            attention_forward2(out, inp, B, T, C, NH, block_size);
            break;
        case 3:
            attention_forward3(out, vaccum, qkvr, preatt, att, inp, B, T, C, NH, block_size);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}

// ----------------------------------------------------------------------------
// random utils

float* make_random_float(int N) {
    float* arr = (float*)malloc(N * sizeof(float));
    for (int i = 0; i < N; i++) {
        arr[i] = ((float)rand() / RAND_MAX) * 2.0 - 1.0;
    }
    return arr;
}

// ----------------------------------------------------------------------------

int main(int argc, char **argv) {
    srand(0);

    int B = 8;
    int T = 1024;
    int C = 768;
    int NH = 12;

    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));

    // create host memory of random numbers
    float* out = (float*)malloc(B * T * C * sizeof(float));
    float* preatt = (float*)malloc(B * NH * T * T * sizeof(float));
    float* att = (float*)malloc(B * NH * T * T * sizeof(float));
    float* inp = make_random_float(B * T * 3 * C);

    // move to GPU
    float* d_out;
    float* d_vaccum;
    float* d_qkvr;
    float* d_preatt;
    float* d_att;
    float* d_inp;
    cudaCheck(hipMalloc(&d_out, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_vaccum, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_qkvr, B * T * 3 * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_preatt, B * NH * T * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_att, B * NH * T * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * T * 3 * C * sizeof(float)));
    cudaCheck(hipMemcpy(d_inp, inp, B * T * 3 * C * sizeof(float), hipMemcpyHostToDevice));

    // read kernel_num from command line
    int kernel_num = 1;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    // first check the correctness of the kernel
    attention_forward_cpu(out, preatt, att, inp, B, T, C, NH);
    attention_forward(kernel_num, d_out, d_vaccum, d_qkvr, d_preatt, d_att, d_inp, B, T, C, NH, 256);

    // compare the output
    float* out_gpu = (float*)malloc(B * T * C * sizeof(float));
    cudaCheck(hipMemcpy(out_gpu, d_out, B * T * C * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < B * T * C; i++) {
        // print the first few comparisons
        if (i < 5) {
            printf("%f %f\n", out[i], out_gpu[i]);
        }
        // ensure correctness for all elements
        if (fabs(out[i] - out_gpu[i]) > 1e-4) {
            printf("Mismatch at %d: %f vs %f\n", i, out[i], out_gpu[i]);
            exit(1);
        }
    }
    printf("Results match!\n");

    // time the kernel at different block sizes
    int block_sizes[] = {32, 64, 128, 256, 512};

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];

        int repeat_times = 10;
        hipEvent_t start, stop;
        cudaCheck(hipEventCreate(&start));
        cudaCheck(hipEventCreate(&stop));
        cudaCheck(hipEventRecord(start, 0));
        for (int i = 0; i < repeat_times; i++) {
            attention_forward(kernel_num, d_out, d_vaccum, d_qkvr, d_preatt, d_att, d_inp, B, T, C, NH, block_size);
        }
        cudaCheck(hipEventRecord(stop, 0));
        cudaCheck(hipEventSynchronize(start));
        cudaCheck(hipEventSynchronize(stop));
        float elapsed_time;
        cudaCheck(hipEventElapsedTime(&elapsed_time, start, stop));

        printf("block_size %4d | time %f ms\n", block_size, elapsed_time);
    }

    // free memory
    free(out);
    free(preatt);
    free(att);
    free(inp);
    free(out_gpu);
    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_vaccum));
    cudaCheck(hipFree(d_qkvr));
    cudaCheck(hipFree(d_preatt));
    cudaCheck(hipFree(d_att));
    cudaCheck(hipFree(d_inp));

    return 0;
}