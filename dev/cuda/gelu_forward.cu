/*
Kernels for gelu forward pass.

Compile example:
nvcc -O3 --use_fast_math gelu_forward.cu -o gelu_forward

version 1 is naive port from CPU code to kernel
./gelu_forward 1
*/

/*
If encountering "error: identifier "M_PI" is undefined", add the following lines to the top of the file:

#define _USE_MATH_DEFINES
#include <math.h>  OR  #include <cmath>

*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// ----------------------------------------------------------------------------
// CUDA utils

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

// error checking
void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};
#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

// ----------------------------------------------------------------------------
// CPU code reference

void gelu_forward_cpu(float* out, float* inp, int N) {
    float s = sqrtf(2.0f / M_PI);
    for (int i = 0; i < N; i++) {
        float x = inp[i];
        float cube = 0.044715f * x * x * x;
        out[i] = 0.5f * x * (1.0f + tanhf(s * (x + cube)));
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

// elementwise ops are nice and ez
__global__ void gelu_kernel(float* out, const float* inp, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float s = sqrtf(2.0f / M_PI);
    if (i < N) {
        float xi = inp[i];
        float cube = 0.044715f * xi * xi * xi;
        out[i] = 0.5f * xi * (1.0f + tanhf(s * (xi + cube)));
    }
}

// ----------------------------------------------------------------------------
// kernel launcher

void gelu_forward1(float* out, float* inp, int N, const int block_size) {
    const int grid_size = CEIL_DIV(N, block_size);
    gelu_kernel<<<grid_size, block_size>>>(out, inp, N);
    cudaCheck(hipGetLastError());
}

// kernel version dispatch
void gelu_forward(int kernel_num,
                  float* out,
                  float* inp,
                  int B, int T, int C,
                  int block_size) {
    switch (kernel_num) {
        case 1:
            gelu_forward1(out, inp, B * T * C, block_size);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}

// ----------------------------------------------------------------------------
// random utils

float* make_random_float(int N) {
    float* arr = (float*)malloc(N * sizeof(float));
    for (int i = 0; i < N; i++) {
        arr[i] = ((float)rand() / RAND_MAX) * 2.0 - 1.0;
    }
    return arr;
}

// ----------------------------------------------------------------------------

int main(int argc, char **argv) {
    srand(0);

    int B = 8;
    int T = 1024;
    int C = 768;

    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));

    // create host memory of random numbers
    float* out = (float*)malloc(B * T * C * sizeof(float));
    float* inp = make_random_float(B * T * C);

    // move to GPU
    float* d_out;
    float* d_inp;
    cudaCheck(hipMalloc(&d_out, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * T * C * sizeof(float)));
    cudaCheck(hipMemcpy(d_inp, inp, B * T * C * sizeof(float), hipMemcpyHostToDevice));

    // read kernel_num from command line
    int kernel_num = 1;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    // first check the correctness of the kernel
    gelu_forward_cpu(out, inp, B * T * C);
    gelu_forward(kernel_num, d_out, d_inp, B, T, C, 128);
    float* out_gpu = (float*)malloc(B * T * C * sizeof(float));
    cudaCheck(hipMemcpy(out_gpu, d_out, B * T * C * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < B * T * C; i++) {
        // print the first few comparisons
        if (i < 5) {
            printf("%f %f\n", out[i], out_gpu[i]);
        }
        // ensure correctness for all elements
        if (fabs(out[i] - out_gpu[i]) > 1e-5) {
            printf("Mismatch at %d: %f vs %f\n", i, out[i], out_gpu[i]);
            exit(1);
        }
    }
    printf("Results match!\n");

    // time the kernel at different block sizes
    int block_sizes[] = {32, 64, 128, 256, 512, 1024};

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];

        int repeat_times = 1000;
        hipEvent_t start, stop;
        cudaCheck(hipEventCreate(&start));
        cudaCheck(hipEventCreate(&stop));
        cudaCheck(hipEventRecord(start, 0));
        for (int i = 0; i < repeat_times; i++) {
            gelu_forward(kernel_num, d_out, d_inp, B, T, C, block_size);
        }
        cudaCheck(hipEventRecord(stop, 0));
        cudaCheck(hipEventSynchronize(start));
        cudaCheck(hipEventSynchronize(stop));
        float elapsed_time;
        cudaCheck(hipEventElapsedTime(&elapsed_time, start, stop));

        // napkin math: estimate the memory bandwidth achieved
        // for each (B,T,C) output element, we do 1 read and 1 write, 4 bytes each
        // and e.g. A100 40GB PCIe is advertised at 1,555GB/s
        long memory_ops = B * T * C * 2 * 4;
        float memory_bandwidth = memory_ops / (elapsed_time / repeat_times) / 1e6;

        printf("block_size %4d | time %f ms | bandwidth %f GB/s\n", block_size, elapsed_time / repeat_times, memory_bandwidth);
    }

    // free memory
    free(out);
    free(inp);
    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_inp));

    return 0;
}