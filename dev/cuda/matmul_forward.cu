/*
Kernels for matmul forward pass.
It's advised to use OpenMP here because the CPU implementation is fairly slow otherwise

Compile example:
nvcc -O3 --use_fast_math -Xcompiler -fopenmp matmul_forward.cu -o matmul_forward -lcublas

version 1 is naive port from CPU code to kernel: parallelizes over B,T, loops over C
OMP_NUM_THREADS=32 ./matmul_forward 1

version 2 calls cuBLAS, very fast
OMP_NUM_THREADS=32 ./matmul_forward 2
*/

#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <omp.h>

// ----------------------------------------------------------------------------
// CUDA utils

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

// error checking
void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};
#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

// ----------------------------------------------------------------------------
// CPU code reference

void matmul_forward_cpu(float* out,
                    float* inp, float* weight, float* bias,
                    int B, int T, int C, int OC) {
    // OC is short for "output channels"
    // inp is (B,T,C), weight is (OC, C), bias is (OC)
    // out will be (B,T,OC)
    #pragma omp parallel for collapse(2)
    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            float* out_bt = out + b * T * OC + t * OC;
            float* inp_bt = inp + b * T * C + t * C;
            for (int o = 0; o < OC; o++) {
                float val = (bias != NULL) ? bias[o] : 0.0f;
                float* wrow = weight + o*C;
                for (int i = 0; i < C; i++) {
                    val += inp_bt[i] * wrow[i];
                }
                out_bt[o] = val;
            }
        }
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

// kernel 1: naive kernel, every thread handles one output element, direct global memory access
__global__ void matmul_forward_kernel1(float* out,
                                       float* inp, float* weight, float* bias,
                                       int BT, int C, int OC) {
    // out is (B,T,OC). OC is short for "output channels", e.g. OC = 4 * C
    // inp is (B,T,C), weight is (OC, C), bias is (OC)
    // in the naive kernel, every thread handles one element of out
    int bt = blockIdx.x * blockDim.x + threadIdx.x;
    int oc = blockIdx.y * blockDim.y + threadIdx.y;
    if (bt < BT && oc < OC) {
        int b = bt / BT;
        int t = bt % BT;
        float val = (bias != NULL) ? bias[oc] : 0.0f;
        float* wrow = weight + oc*C;
        float* inp_bt = inp + b * BT * C + t * C;
        for (int i = 0; i < C; i++) {
            val += inp_bt[i] * wrow[i];
        }
        out[bt * OC + oc] = val;
    }
}

// is there no better way other than just adding bias with a whole separate kernel?
// this is a highly memory-bound operation, should be fused into the matmul kernel
// but i can't seem to find a cuBLAS function that does this
__global__ void add_bias(float* out, float* bias, int B, int T, int OC) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < B * T * OC; i += stride) {
        int col = i % OC;
        out[i] += bias[col];
    }
}

// ----------------------------------------------------------------------------
// kernel launcher

// kernel 1 is the most naive matmul kernel
void matmul_forward1(float* out,
                     float* inp, float* weight, float* bias,
                     int B, int T, int C, int OC,
                     const int sqrt_block_size) {
    // out is (B,T,OC). OC is short for "output channels", e.g. OC = 4 * C
    // inp is (B,T,C), weight is (OC, C), bias is (OC)
    dim3 gridDim(CEIL_DIV(B * T, sqrt_block_size), CEIL_DIV(OC, sqrt_block_size));
    dim3 blockDim(sqrt_block_size, sqrt_block_size);
    matmul_forward_kernel1<<<gridDim, blockDim>>>(out, inp, weight, bias, B*T, C, OC);
    cudaCheck(hipGetLastError());
}

// kernel 2 calls cuBLAS, which should be very efficient
void matmul_forward2(float* out,
                     float* inp, float* weight, float* bias,
                     int B, int T, int C, int OC,
                     const int sqrt_block_size) {
    hipblasHandle_t handle; // cuBLAS context
    hipblasStatus_t stat = hipblasCreate(&handle); // initialize CUBLAS context
    // for reference API is:
    // cublasStatus_t cublasSgemm(cublasHandle_t handle,
    //                        cublasOperation_t transa, cublasOperation_t transb,
    //                        int m, int n, int k,
    //                        const float           *alpha,
    //                        const float           *A, int lda,
    //                        const float           *B, int ldb,
    //                        const float           *beta,
    //                        float           *C, int ldc)
    // for us, inp is (B*T, C), weight is (OC, C), out is (B*T, OC)
    // cuBLAS does C = alpha * A * B + beta * C
    // where A is mxk, B is kxn, C is mxn
    // now, because we use row-major storage, cuBLAS (which is column-major) sees our matrices transposed.
    // algorithmically / in e.g. PyTorch we want to do: out = inp @ weight.T
    // but because cuBLAS is column-major, we actually want to get it to calculate out.T . Mathematically, this is:
    // out.T = weight @ inp.T
    // but again, our variables look transposed, so using the actual weight/inp we have here in this function, this becomes
    // out.T = weight.T @ inp
    // so we need to get cuBLAS to calculate weight.T @ inp (the variables here are the actual ones in this function)
    // => need to call cuBLAS with A = weight, B = inp
    // => need to call cuBLAS with transa = CUBLAS_OP_T, transb = CUBLAS_OP_N

    const float alpha = 1.0f;
    const float beta = 0.0f;
    stat = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, OC, B*T, C, &alpha, weight, C, inp, C, &beta, out, OC);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("hipblasSgemm failed\n");
        exit(1);
    }
    // and now we still have to add the bias... (ew)
    if (bias != NULL) {
        int block_size = sqrt_block_size * sqrt_block_size;
        int grid_size = CEIL_DIV(OC * B * T, block_size);
        add_bias<<<grid_size, block_size>>>(out, bias, B, T, OC);
        cudaCheck(hipGetLastError());
    }
    hipblasDestroy(handle);
}

// kernel version dispatch
void matmul_forward(int kernel_num,
                    float* out,
                    float* inp, float* weight, float* bias,
                    int B, int T, int C, int OC,
                    const int sqrt_block_size) {
    switch (kernel_num) {
        case 1:
            matmul_forward1(out, inp, weight, bias, B, T, C, OC, sqrt_block_size);
            break;
        case 2:
            matmul_forward2(out, inp, weight, bias, B, T, C, OC, sqrt_block_size);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}

// ----------------------------------------------------------------------------
// random utils

float* make_random_float(int N) {
    float* arr = (float*)malloc(N * sizeof(float));
    for (int i = 0; i < N; i++) {
        arr[i] = ((float)rand() / RAND_MAX) * 2.0 - 1.0;
    }
    return arr;
}

// ----------------------------------------------------------------------------

int main(int argc, char **argv) {
    srand(0);

    int B = 8;
    int T = 1024;
    int C = 768;
    int OC = 768 * 4; // expansion of 4, e.g. in the MLP

    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));

    // create host memory of random numbers
    float* out = (float*)malloc(B * T * OC * sizeof(float));
    float* inp = make_random_float(B * T * C);
    float* weight = make_random_float(OC * C);
    float* bias = make_random_float(OC);

    // move to GPU
    float* d_out;
    float* d_inp;
    float* d_weight;
    float* d_bias;
    cudaCheck(hipMalloc(&d_out, B * T * OC * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_weight, C * OC * sizeof(float)));
    cudaCheck(hipMalloc(&d_bias, OC * sizeof(float)));
    cudaCheck(hipMemcpy(d_inp, inp, B * T * C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_weight, weight, C * OC * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_bias, bias, OC * sizeof(float), hipMemcpyHostToDevice));

    // read kernel_num from command line
    int kernel_num = 1;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    // first check the correctness of the kernel
    matmul_forward_cpu(out, inp, weight, bias, B, T, C, OC);
    matmul_forward(kernel_num, d_out, d_inp, d_weight, d_bias, B, T, C, OC, 32);

    float* out_gpu = (float*)malloc(B * T * OC * sizeof(float));
    cudaCheck(hipMemcpy(out_gpu, d_out, B * T * OC * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < B * T * OC; i++) {
        // print the first few comparisons
        if (i < 5) {
            printf("%f %f\n", out[i], out_gpu[i]);
        }
        // ensure correctness for all elements
        if (fabs(out[i] - out_gpu[i]) > 1e-4) {
            printf("Mismatch at %d: %f vs %f\n", i, out[i], out_gpu[i]);
            exit(1);
        }
    }
    printf("Results match at block_size=1024!\n");

    // time the kernel at different block sizes
    int sqrt_block_sizes[] = {4, 8, 16, 32};

    for (int j = 0; j < sizeof(sqrt_block_sizes) / sizeof(int); j++) {
        int sqrt_block_size = sqrt_block_sizes[j];

        int repeat_times = 10;
        hipEvent_t start, stop;
        cudaCheck(hipEventCreate(&start));
        cudaCheck(hipEventCreate(&stop));
        cudaCheck(hipEventRecord(start, 0));
        for (int i = 0; i < repeat_times; i++) {
            matmul_forward(kernel_num, d_out, d_inp, d_weight, d_bias, B, T, C, OC, sqrt_block_size);
        }
        cudaCheck(hipEventRecord(stop, 0));
        cudaCheck(hipEventSynchronize(start));
        cudaCheck(hipEventSynchronize(stop));
        float elapsed_time;
        cudaCheck(hipEventElapsedTime(&elapsed_time, start, stop));

        // napkin math: estimate the flops achieved
        // e.g. A100 40GB PCIe is advertised at 19.5 TFLOPS fp32
        float tflops = (float)B * T * C * OC * 2 * repeat_times / elapsed_time * 1e3f / 1e12f;
        printf("sqrt_block_size %4d | time %f ms | tflops %f\n", sqrt_block_size, elapsed_time, tflops);
    }

    // free memory
    free(out);
    free(inp);
    free(weight);
    free(bias);
    free(out_gpu);
    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_inp));
    cudaCheck(hipFree(d_weight));
    cudaCheck(hipFree(d_bias));

    return 0;
}