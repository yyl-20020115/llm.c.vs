/*
Kernels for crossentropy forward pass.

Compile example:
nvcc -O3 --use_fast_math crossentropy_forward.cu -o crossentropy_forward

version 1 is a straight-forward port from CPU code to kernel, parallel over B,T
./crossentropy_forward 1
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// ----------------------------------------------------------------------------
// CUDA utils

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

// error checking
void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};
#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

// ----------------------------------------------------------------------------
// CPU code reference

void crossentropy_forward_cpu(float* losses,
                            float* probs, int* targets,
                            int B, int T, int V) {
    // output: losses is (B,T) of the individual losses at each position
    // input: probs are (B,T,V) of the probabilities
    // input: targets is (B,T) of integers giving the correct index in logits
    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            // loss = -log(probs[target])
            float* probs_bt = probs + b * T * V + t * V;
            int ix = targets[b * T + t];
            losses[b * T + t] = -logf(probs_bt[ix]);
        }
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

__global__ void crossentropy_forward_kernel1(float* losses,
                            float* probs, int* targets,
                            int B, int T, int V) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < B * T) {
        int b = i / T;
        int t = i % T;
        float* probs_bt = probs + b * T * V + t * V;
        int ix = targets[b * T + t];
        losses[b * T + t] = -logf(probs_bt[ix]);
    }
}

// ----------------------------------------------------------------------------
// kernel launcher

void crossentropy_forward1(float* losses,
                            float* probs, int* targets,
                            int B, int T, int V,
                            const int block_size) {
    const int N = B * T;
    const int grid_size = CEIL_DIV(N, block_size);
    crossentropy_forward_kernel1<<<grid_size, block_size>>>(losses, probs, targets, B, T, V);
    cudaCheck(hipGetLastError());
}

// kernel version dispatch
void crossentropy_forward(int kernel_num,
                            float* losses,
                            float* probs, int* targets,
                            int B, int T, int V,
                            const int block_size) {
    switch (kernel_num) {
        case 1:
            crossentropy_forward1(losses, probs, targets, B, T, V, block_size);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}

// ----------------------------------------------------------------------------
// random utils

float* make_random_float(int N) {
    float* arr = (float*)malloc(N * sizeof(float));
    for (int i = 0; i < N; i++) {
        arr[i] = ((float)rand() / RAND_MAX); // [0,1)
    }
    return arr;
}

int* make_random_int(int N, int V) {
    int* arr = (int*)malloc(N * sizeof(int));
    for (int i = 0; i < N; i++) {
        arr[i] = rand() % V;
    }
    return arr;
}

// ----------------------------------------------------------------------------

int main(int argc, char **argv) {
    srand(0);

    int B = 8;
    int T = 1024;
    int V = 50257;

    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));

    // create host memory of random numbers
    float* out = (float*)malloc(B * T * sizeof(float));
    float* probs = make_random_float(B * T * V);
    int* targets = make_random_int(B * T, V);

    // move to GPU
    float* d_out;
    float* d_probs;
    int* d_targets;
    cudaCheck(hipMalloc(&d_out, B * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_probs, B * T * V * sizeof(float)));
    cudaCheck(hipMalloc(&d_targets, B * T * sizeof(int)));
    cudaCheck(hipMemcpy(d_probs, probs, B * T * V * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_targets, targets, B * T * sizeof(int), hipMemcpyHostToDevice));

    // read kernel_num from command line
    int kernel_num = 1;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    // first check the correctness of the kernel
    crossentropy_forward_cpu(out, probs, targets, B, T, V);
    crossentropy_forward(kernel_num, d_out, d_probs, d_targets, B, T, V, 256);
    float* out_gpu = (float*)malloc(B * T * sizeof(float));
    cudaCheck(hipMemcpy(out_gpu, d_out, B * T * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < B * T; i++) {
        // print the first few comparisons
        if (i < 10) {
            printf("%f %f\n", out[i], out_gpu[i]);
        }
        // ensure correctness for all elements
        if (fabs(out[i] - out_gpu[i]) > 1e-5) {
            printf("Mismatch at %d: %f vs %f\n", i, out[i], out_gpu[i]);
            exit(1);
        }
    }
    printf("Results match at block_size=256!\n");

    // time the kernel at different block sizes
    int block_sizes[] = {32, 64, 128, 256, 512, 1024};

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];

        int repeat_times = 1000;
        hipEvent_t start, stop;
        cudaCheck(hipEventCreate(&start));
        cudaCheck(hipEventCreate(&stop));
        cudaCheck(hipEventRecord(start, 0));
        for (int i = 0; i < repeat_times; i++) {
            crossentropy_forward(kernel_num, d_out, d_probs, d_targets, B, T, V, block_size);
        }
        cudaCheck(hipEventRecord(stop, 0));
        cudaCheck(hipEventSynchronize(start));
        cudaCheck(hipEventSynchronize(stop));
        float elapsed_time;
        cudaCheck(hipEventElapsedTime(&elapsed_time, start, stop));

        printf("block_size %4d | time %f ms\n", block_size, elapsed_time / repeat_times);
    }

    // free memory
    free(out);
    free(probs);
    free(targets);
    free(out_gpu);
    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_probs));
    cudaCheck(hipFree(d_targets));

    return 0;
}